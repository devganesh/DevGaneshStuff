#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
extern "C"
{
__global__ void truncnorm_kernel(float *x, int n, float *mu, float *sigma,
float *a, float *b,int rng_a, int rng_b, int rng_c,int maxtries)
{
  // Usual block/thread indexing... 
  int myblock = blockIndex.x + blockIndex.y * gridDim.x;
  int blocksize = blockDim.x * blockDim.y * blockDim.z;
  int subthread = threadIndex.z*(blockDim.x * blockDim.y) + threadIndex.y*blockDim.x + threadIndex.x;
  int index = myblock * blocksize + subthread;
  hiprandState rng;
  hiprand_init(rng_a + index*rng_b, rng_c, 0, &rng);                                                                           
  int numtries = 1;
  float uExp, alpha, z, psi, u;
  
  if(index>n) return;

  // Rejection sampling 
  while(numtries <= maxtries)
  {
    x[index] = mu[index] + sigma[index]*hiprand_normal(&rng);
    if( x[index] >= a[index] && x[index] <= b[index]) return;
    numtries++;
  }
  //If all samples rejected, use methods given in Robert(2009)  
  while(1)
  {
    // for the case when the lower limit of the truncation region is not finite
    if(!isfinite(a[index]))
    {
      a[index] = -b[index];
      b[index] = HIP_NAN_F;
    }
    uExp = hiprand_uniform(&rng);                
    alpha = (a[index] + sqrt(a[index]*a[index] + 4))/2;
    z = a[index] - (log(uExp)/alpha);
    
    if(a[index] < alpha)  psi = exp(-(alpha - z)*(alpha - z)/2);
    else  psi = exp(-(alpha - z)*(alpha - z)/2)*exp(-(a[index] - alpha)*(a[index] - alpha)/2);
    u = hiprand_uniform(&rng);
    if(u < psi)
    {
      x[index] = sigma[index]*z + mu[index];
      if(!isfinite(a[index])) x[index] = -x[index];
      return;
    }
  }     return;}} // END extern "C"
